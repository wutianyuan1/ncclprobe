#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>


#define N_REPEAT 3
#define SNEDPEER 3
#define RECVPEER 2

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

static uint64_t getIDHash(const char* ID) {
  return getHostHash(ID);
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


__global__ void fillBuffer(float *buf, int value, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        buf[idx] = value;
    }
}



int main(int argc, char* argv[])
{
  int size = 128 * 1024 * 1024;
  int myRank, nRanks, localRank = 0;

  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  auto rank_str = std::to_string(myRank);
  setenv("RANK", rank_str.c_str(), 1);
  setenv("LOCAL_RANK", rank_str.c_str(), 1);

  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }
  printf("[Rank %d] localRank=%d\n", myRank, localRank);

  // Define groups
  int group = 0;// myRank / 2; // This will be 0 for ranks 0,1 and 1 for ranks 2,3

  // Create MPI sub-communicators based on groups
  MPI_Comm subComm;
  MPICHECK(MPI_Comm_split(MPI_COMM_WORLD, group, myRank, &subComm));

  int subRank, subSize;
  MPICHECK(MPI_Comm_rank(subComm, &subRank));
  MPICHECK(MPI_Comm_size(subComm, &subSize));

  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  // Get NCCL unique ID at sub-communicator rank 0 and broadcast it to all others in the sub-communicator
  if (subRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, subComm));
  printf("[Rank %d], subRank=%d, subSize=%d, unique ID=%lu\n", myRank, subRank, subSize, getIDHash(id.internal));

  // Picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  fillBuffer<<<blocksPerGrid, threadsPerBlock, 0, s>>>(sendbuff, myRank, size);
  fillBuffer<<<blocksPerGrid, threadsPerBlock, 0, s>>>(recvbuff, 10086, size);
  CUDACHECK(hipStreamSynchronize(s));

  // Initializing NCCL with sub-communicator
  NCCLCHECK(ncclCommInitRank(&comm, subSize, id, subRank));
  MPI_Barrier(MPI_COMM_WORLD);

  // Communicating using NCCL within sub-communicators
  // int nn = myRank == 1 ? N_REPEAT : N_REPEAT - 1;
  for (int i = 0; i < N_REPEAT; i++) {
    // hipEvent_t start, stop;
    // float duration;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    // if (myRank == SNEDPEER) {
    //   sleep(1);
    //   hipEventRecord(start, s);
    //   NCCLCHECK(ncclSend(sendbuff, size, ncclFloat, RECVPEER, comm, s));
    //   hipEventRecord(stop, s);
      
    // } else if (myRank == RECVPEER) {
    //   hipEventRecord(start, s);
    //   NCCLCHECK(ncclRecv(recvbuff, size, ncclFloat, SNEDPEER, comm, s));
    //   hipEventRecord(stop, s);
    // } else {
    //   hipEventRecord(start, s);
    //   hipEventRecord(stop, s);
    // }
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&duration, start, stop);
    // printf("Rank %d, time: %f\n", myRank, duration);

    // NCCLCHECK(ncclGroupStart());
    NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclAvg, comm, s));
    // NCCLCHECK(ncclGroupEnd());
    printf("Rank %d, allreduce: %d\n", myRank, i);
  }

  // Completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  float* cpubuff = (float*)malloc(sizeof(float) * size);
  hipMemcpy(cpubuff, recvbuff, sizeof(float)*size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("[Rank %d]: result=%f\n", myRank, cpubuff[0]);

  // Free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));

  // Finalizing NCCL
  ncclCommDestroy(comm);

  while (1) {};

  // Finalizing MPI sub-communicator
  MPICHECK(MPI_Comm_free(&subComm));

  // Finalizing MPI
  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}
